#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
clock_t sum_time = 0;
const int TILE = 16;
const int SIDE = 4;
__global__ void kernel(int *or_mat, int * to){
	__shared__ int mat[TILE][TILE + 1];
	int x = blockIdx.x * TILE + threadIdx.x;
	int y = blockIdx.y * TILE + threadIdx.y;
#pragma unroll
	for (int k = 0; k < TILE; k += SIDE) {
		if (x < 1024 && y + k < 1024)mat[threadIdx.y + k][threadIdx.x] = or_mat[((y + k) * 1024) + x];
	}
	__syncthreads();
	x = blockIdx.y * TILE + threadIdx.x;
	y = blockIdx.x * TILE + threadIdx.y;
#pragma unroll
	for (int k = 0; k < TILE; k += SIDE){
		if (x < 1024 && y + k < 1024)to[(y + k) * 1024 + x] = mat[threadIdx.x][threadIdx.y + k];
	}
}
int main(){
	int n, it, i, j;
	int * mat, *to, *d_mat, *d_to;
	int *d_mat2, *d_to2;
	int *d_mat3, *d_to3;
	int *d_mat4, *d_to4;
	n = 1024;
	mat = (int*)malloc(n*n*sizeof(int));
	to = (int*)malloc(n*n*sizeof(int));
	printf("input the iter times:\n");
	scanf("%d", &it);
	for (i = 0; i<n; i++){
		for (j = 0; j<n; j++){
			mat[i*n + j] = 1;
		}
	}
	hipMalloc((void**)&d_mat, n*n*sizeof(int));
	hipMalloc((void**)&d_to, n*n*sizeof(int));
	hipMemcpy(d_mat, mat, n*n*sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_mat2, n*n*sizeof(int));
	hipMalloc((void**)&d_to2, n*n*sizeof(int));
	hipMemcpy(d_mat2, mat, n*n*sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_mat3, n*n*sizeof(int));
	hipMalloc((void**)&d_to3, n*n*sizeof(int));
	hipMemcpy(d_mat3, mat, n*n*sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_mat4, n*n*sizeof(int));
	hipMalloc((void**)&d_to4, n*n*sizeof(int));
	hipMemcpy(d_mat4, mat, n*n*sizeof(int), hipMemcpyHostToDevice);
	dim3 dimBlock(16, 4);
	dim3 dimGrid(64, 64);
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	hipStream_t stream[4];
	for (i = 0; i < 4; i++){
		hipStreamCreate(&stream[i]);
	}
	for (i = 0; i<it; i++){
		kernel << <dimGrid, dimBlock,sizeof(int)*TILE*(TILE+1),stream[0] >> >(d_mat, d_to);
		kernel << <dimGrid, dimBlock, sizeof(int)*TILE*(TILE + 1), stream[1] >> >(d_mat2, d_to2);
		kernel << <dimGrid, dimBlock, sizeof(int)*TILE*(TILE + 1), stream[2] >> >(d_mat3, d_to3);
		kernel << <dimGrid, dimBlock, sizeof(int)*TILE*(TILE + 1), stream[3] >> >(d_mat4, d_to4);
		hipDeviceSynchronize();
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("The total running time is: %f\n", time);
    printf("input any to exit\n");
	scanf("%d", &it);
}
