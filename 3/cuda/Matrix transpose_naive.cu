#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
clock_t sum_time = 0;
__global__ void kernel(int *mat, int * to){
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int index_in = i * 1024 + j;
	int index_out = j * 1024 + i;
	to[index_out] = mat[index_in];
}
int main(){
	int n, it, i, j;
	int * mat, *to, *d_mat, *d_to;
	n = 1024;
	mat = (int*)malloc(n*n*sizeof(int));
	to = (int*)malloc(n*n*sizeof(int));
	printf("input the iter times:\n");
	scanf("%d", &it);
	for (i = 0; i<n; i++){
		for (j = 0; j<n; j++){
			mat[i*n + j] = 1;
		}
	}
	hipMalloc((void**)&d_mat, n*n*sizeof(int));
	hipMalloc((void**)&d_to, n*n*sizeof(int));
	hipMemcpy(d_mat, mat, n*n*sizeof(int), hipMemcpyHostToDevice);
	dim3 dimBlock(16, 16);
	dim3 dimGrid(64, 64);
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	for (i = 0; i<it; i++){
		kernel <<<dimGrid, dimBlock>>>(d_mat, d_to);
	}
	hipEventRecord(stop,0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("The total running time is: %f\n", time);
	scanf("%d", &it);
}