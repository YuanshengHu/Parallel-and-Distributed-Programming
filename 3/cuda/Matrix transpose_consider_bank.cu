#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
clock_t sum_time = 0;
__global__ void kernel(int *or_mat, int * to){
	__shared__ int mat[16][17];
	int bx = blockIdx.x * 16;
	int by = blockIdx.y * 16;
	int i = by + threadIdx.y; int j = bx + threadIdx.x;
	int ti = bx + threadIdx.y; int tj = by + threadIdx.x;
	if (i < 1024 && j < 1024){
		mat[threadIdx.x][threadIdx.y] = or_mat[i * 1024 + j];
	}
	__syncthreads();
	if (tj < 1024 && ti < 1024){
		to[ti * 1024 + tj] = mat[threadIdx.y][threadIdx.x];
	}
}
int main(){
	int n, it, i, j;
	int * mat, *to, *d_mat, *d_to;
	n = 1024;
	mat = (int*)malloc(n*n*sizeof(int));
	to = (int*)malloc(n*n*sizeof(int));
	printf("input the iter times:\n");
	scanf("%d", &it);
	for (i = 0; i<n; i++){
		for (j = 0; j<n; j++){
			mat[i*n + j] = 1;
		}
	}
	hipMalloc((void**)&d_mat, n*n*sizeof(int));
	hipMalloc((void**)&d_to, n*n*sizeof(int));
	hipMemcpy(d_mat, mat, n*n*sizeof(int), hipMemcpyHostToDevice);
	dim3 dimBlock(16, 16);
	dim3 dimGrid(64, 64);
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	for (i = 0; i<it; i++){
		kernel << <dimGrid, dimBlock >> >(d_mat, d_to);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("The total running time is: %f\n", time);
    printf("input any to exit\n");
	scanf("%d", &it);
}
