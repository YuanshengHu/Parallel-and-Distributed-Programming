#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
clock_t sum_time = 0;
const int TILE = 32;
const int SIDE = 8;
__global__ void kernel(int *or_mat, int * to){
	__shared__ int mat[TILE][TILE + 1];
	int x = blockIdx.x * TILE + threadIdx.x;
	int y = blockIdx.y * TILE + threadIdx.y;
#pragma unroll
	for (int k = 0; k < TILE; k += SIDE) {
		if (x < 2048 && y + k < 2048)mat[threadIdx.y + k][threadIdx.x] = or_mat[((y + k) * 2048) + x];
	}
	__syncthreads();
	x = blockIdx.y * TILE + threadIdx.x;
	y = blockIdx.x * TILE + threadIdx.y;
#pragma unroll
	for (int k = 0; k < TILE; k += SIDE){
		if (x < 2048 && y + k < 2048)to[(y + k) * 2048 + x] = mat[threadIdx.x][threadIdx.y + k];
	}
}
int main(){
	int n, it, i, j;
	int * mat, *to, *d_mat, *d_to;
	int *d_mat2, *d_to2;
	int *d_mat3, *d_to3;
	int *d_mat4, *d_to4;
	n = 2048;
	mat = (int*)malloc(n*n*sizeof(int));
	to = (int*)malloc(n*n*sizeof(int));
	printf("input the iter times:\n");
	scanf("%d", &it);
	for (i = 0; i<n; i++){
		for (j = 0; j<n; j++){
			mat[i*n + j] = 1;
		}
	}
	hipMalloc((void**)&d_mat, n*n*sizeof(int));
	hipMalloc((void**)&d_to, n*n*sizeof(int));
	hipMemcpy(d_mat, mat, n*n*sizeof(int), hipMemcpyHostToDevice);
	dim3 dimBlock(32, 8);
	dim3 dimGrid(64, 64);
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	for (i = 0; i<it; i++){
		kernel << <dimGrid, dimBlock>> >(d_mat, d_to);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("The total running time is: %f\n", time);
    printf("input any to exit\n");
	scanf("%d", &it);
}
